#include <hip/hip_runtime.h>

#include <hipfft/hipfft.h>

#include <complex>
#include <hip/device_functions.h>
#include <hip/hip_complex.h>
#include <chrono>
#include <iostream>
#pragma comment(lib,"cufft.lib")

using namespace std;

__global__
void Complex_mult(hipfftComplex * c, const hipfftComplex * a, const hipfftComplex * b) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    c[i].x = (a[i].x * b[i].x - a[i].y * (-b[i].y));
    c[i].y = (a[i].x * (-b[i].y) + a[i].y * b[i].x);
}

std::chrono::time_point<std::chrono::high_resolution_clock> now()
{
    return std::chrono::high_resolution_clock::now();
}

template <typename T>
double milliseconds(T t)
{
    return (double) std::chrono::duration_cast<std::chrono::nanoseconds>(t).count() / 1000000;
}

extern "C"
hipfftComplex* FFT_GPU(hipfftComplex* signal1, hipfftComplex* signal2, int len_c)
{
    hipfftComplex* GPU_data_first;
    hipfftComplex* GPU_data_second;
    auto t1 = now();
    hipMalloc((void**)&GPU_data_first, len_c * sizeof(hipfftComplex));
    hipMalloc((void**)&GPU_data_second, len_c * sizeof(hipfftComplex));
    hipfftHandle plan1;
    hipfftPlan1d(&plan1, len_c, HIPFFT_C2C, 1);


    auto t2 = now();
    cout<<"VIDEO MEM: "<< milliseconds(t2-t1)<<" ms"<<endl;
    hipMemcpy(GPU_data_first, signal1, len_c * sizeof(hipfftComplex), hipMemcpyHostToDevice);
    hipMemcpy(GPU_data_second, signal2, len_c * sizeof(hipfftComplex), hipMemcpyHostToDevice);
    auto t3 = now();
    cout<<"COPY DATA: "<< milliseconds(t3-t2)<<" ms"<<endl;

    auto t3_1 = now();

    //cudaDeviceSynchronize();
    hipfftExecC2C(plan1, (hipfftComplex*)GPU_data_first, (hipfftComplex*)GPU_data_first, HIPFFT_FORWARD);
    hipfftExecC2C(plan1, (hipfftComplex*)GPU_data_second, (hipfftComplex*)GPU_data_second, HIPFFT_FORWARD);
    auto t4 = now();
    cout<<"FFT: "<< milliseconds(t4-t3_1)<<" ms"<<endl;
    //cufftDestroy(plan1); // освобождение памяти

    hipfftComplex* Mult_result;
    hipMalloc((void**)&Mult_result, len_c * sizeof(hipfftComplex));

    auto t6 = now();

    Complex_mult <<<256, 192>>>(Mult_result, GPU_data_first, GPU_data_second);

    auto t7 = now();

    cout<<"cMULT: "<< milliseconds(t7-t6)<<" ms"<<endl;
    hipFree(GPU_data_first);
    hipFree(GPU_data_second);
    hipfftHandle plan3;

    auto t8 = now();
   // cufftPlan1d(&plan3, len_c, CUFFT_C2C, 1);
    hipfftExecC2C(plan1, (hipfftComplex*)Mult_result, (hipfftComplex*)Mult_result, HIPFFT_BACKWARD);
    auto t9 = now();
    cout<<"IFFT: "<< milliseconds(t9-t8)<<" ms"<<endl;

    hipfftComplex* result_of_IFFT = new hipfftComplex[len_c];
    auto t10 = now();
    hipMemcpy(result_of_IFFT, Mult_result, sizeof(hipfftComplex) * (len_c), hipMemcpyDeviceToHost);
    auto t11 = now();
    cout<<"COPY FROM VIDEO: "<< milliseconds(t11-t10)<<" ms"<<endl;
    hipFree(Mult_result);

    return result_of_IFFT;
}
